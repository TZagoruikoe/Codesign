#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumVec(int* vec1, int* vec2, int* res, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        vec1[idx] = idx * 10;
        vec2[idx] = idx * 2;
        res[idx] = vec1[idx] + vec2[idx];
    }
}

int main() {
    int* vec1;
    int* vec2;
    int* res;
    const int data_size = 1024 * 1024;

    hipMalloc(&vec1, data_size * sizeof(int));
    hipMalloc(&vec2, data_size * sizeof(int));
    hipMalloc(&res, data_size * sizeof(int));

    hipEvent_t start, stop, sync_event;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&sync_event);

    hipEventRecord(start);

    sumVec<<<(data_size + 255) / 256, 256>>>(vec1, vec2, res, data_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Kernel execution time: %.2f ms\n", elapsed_time);

    hipEventRecord(sync_event);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    sumVec<<<(data_size + 255) / 256, 256, 0, stream1>>>(vec1, vec2, res, data_size / 2);
    sumVec<<<(data_size + 255) / 256, 256, 0, stream2>>>(vec1 + data_size / 2, vec2 + data_size / 2, res + data_size / 2, data_size / 2);

    hipStreamWaitEvent(stream1, sync_event, 0);
    hipStreamWaitEvent(stream2, sync_event, 0);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    printf("Streams synchronized after sync_event.\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(sync_event);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(vec1);
    hipFree(vec2);
    hipFree(res);

    return 0;
}